#include "hip/hip_runtime.h"
#include "utils.h"

struct sqrtupdateOutput_functor
{
  const float bias;

  sqrtupdateOutput_functor(float bias_) : bias(bias_) {}

  __device__ void operator()(float* output, const float* input) const
  {
    *output = sqrt(*input + bias);
  }
};

static int cunn_Sqrt_updateOutput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  float bias = (float) luaT_getfieldchecknumber(L,1,"eps");
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  THAssert(THCudaTensor_checkGPU(state, 2, input, output));
  THCudaTensor_resizeAs(state, output, input);
  THCudaTensor_pointwiseApply2(state, output, input, sqrtupdateOutput_functor(bias));
  return 1;
}

struct sqrtupdateGradInput_functor
{
  sqrtupdateGradInput_functor() {}

  __device__ void operator()(float* gradInput, const float* output, const float* gradOutput) const
  {
    *gradInput = (*output == 0.0f) ? 0.0f : ((0.5f * *gradOutput) / *output);
  }
};

static int cunn_Sqrt_updateGradInput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  THCudaTensor *gradInput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  THAssert(THCudaTensor_checkGPU(state, 3, output, gradOutput, gradInput));
  THCudaTensor_resizeAs(state, gradInput, output);
  THCudaTensor_pointwiseApply3(state, gradInput, output, gradOutput, sqrtupdateGradInput_functor());
  return 1;
}

static const struct luaL_Reg cunn_Sqrt__ [] = {
  {"Sqrt_updateOutput", cunn_Sqrt_updateOutput},
  {"Sqrt_updateGradInput", cunn_Sqrt_updateGradInput},
  {NULL, NULL}
};

static void cunn_Sqrt_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_Sqrt__, "nn");
  lua_pop(L,1);
}
