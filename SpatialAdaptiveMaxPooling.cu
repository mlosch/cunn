#include "hip/hip_runtime.h"

#define CUDA_MAX_THREADS 1024   // this is safe, in reality 256 is our limit

/*
 * Description:
 *    this function adaptively maxpools an input 4D tensor along dimensions 2 and 3
 *    4D input, 4D output, 4D argmax x and y
 */
__global__ void adaptivemaxpool(float *input, float *output, float *indices_x, float *indices_y,
                        int input_n, int input_h, int input_w,
                        int output_h, int output_w)
{
  // iterators
  int xx, yy;

  // compute offsets based on thread/block ID
  int o = blockIdx.x;
  int i = o;
  //int k = blockIdx.x % input_n;

  int xx_start = threadIdx.x;
  int xx_end = output_w;
  const int xx_step = blockDim.x;

  int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
  int yy_end = output_h;
  const int yy_step = blockDim.y*gridDim.y;

  // select input/output plane
  output = output + o*output_w*output_h;
  input = input + i*input_w*input_h;
  indices_x = indices_x + o*output_w*output_h;
  indices_y = indices_y + o*output_w*output_h;

  // For all output pixels...
  for(yy = yy_start; yy < yy_end; yy+=yy_step) {

    int y_start = (int)floor(float(yy) / output_h * input_h);
    int y_end   = (int)ceil(float(yy+1) / output_h * input_h);
    int kH = y_end-y_start;

    for(xx = xx_start; xx < xx_end; xx+=xx_step) {
      int x_start = (int)floor(float(xx) / output_w * input_w);
      int x_end   = (int)ceil(float(xx + 1) / output_w * input_w);

      int kW = x_end-x_start;

      // Compute the mean of the input image...
      float *ptr_input = input + y_start*input_w + x_start;
      float *ptr_output = output + yy*output_w + xx;
      float *ptr_ind_x = indices_x + yy*output_w + xx;
      float *ptr_ind_y = indices_y + yy*output_w + xx;
      int argmax_x = -1;
      int argmax_y = -1;
      float max = -FLT_MAX;
      int kx, ky;
      for(ky = 0; ky < kH; ky++) {
        for(kx = 0; kx < kW; kx++) {
          float val = ptr_input[kx];
          if (val > max) {
            max = val;
            argmax_x = kx;
            argmax_y = ky;
          }
        }
        ptr_input += input_w; // next input line
      }
      // Update output and argmax
      *ptr_output = max;
      *ptr_ind_x = argmax_x + 1;
      *ptr_ind_y = argmax_y + 1;
    }
  }
}

/*
 * Description:
 *    this function computes the gradInput from weight and gradOutput
 */
__global__ void adaptivemaxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y,
                             int input_n, int input_h, int input_w,
                             int output_h, int output_w)
{
  // iterators
  int xx, yy;

  // compute offsets based on thread/block ID
  int o = blockIdx.x;
  int i = o;
  //int k = blockIdx.x % input_n;

  int xx_start = threadIdx.x;
  int xx_end = output_w;
  int xx_step = blockDim.x;

  int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
  int yy_end = output_h;
  int yy_step = blockDim.y*gridDim.y;

  // select input/output plane
  gradOutput = gradOutput + o*output_w*output_h;
  gradInput = gradInput + i*input_w*input_h;
  indices_x = indices_x + o*output_w*output_h;
  indices_y = indices_y + o*output_w*output_h;

  // compute gradInput
  for(yy = yy_start; yy < yy_end; yy+=yy_step) {

    int y_start = (int)floor(float(yy) / output_h * input_h);

    for(xx = xx_start; xx < xx_end; xx+=xx_step) {

      int x_start = (int)floor(float(xx) / output_w * input_w);

      float *ptr_gradInput = gradInput + y_start*input_w + x_start;
      float *ptr_gradOutput = gradOutput + yy*output_w + xx;
      float *ptr_ind_x = indices_x + yy*output_w + xx;
      float *ptr_ind_y = indices_y + yy*output_w + xx;
      float z = *ptr_gradOutput;

      int argmax_x = (*ptr_ind_x)-1;
      int argmax_y = (*ptr_ind_y)-1;

      ptr_gradInput[argmax_x + argmax_y*input_w] += z;
    }
  }
}

/*
 * Description:
 *    this function computes the gradInput from weight and gradOutput
 *    when kH != dH or kW != dW (uses atomic add)
 */
__global__ void atomicadaptivemaxgradinput(
  float *gradInput, float *gradOutput, float *indices_x, float *indices_y,
  int input_n, int input_h, int input_w, int output_h, int output_w
)
{
  // iterators
  int xx, yy;

  // compute offsets based on thread/block ID
  int o = blockIdx.x;
  int i = o;

  int xx_start = threadIdx.x;
  int xx_end = output_w;
  int xx_step = blockDim.x;

  int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
  int yy_end = output_h;
  int yy_step = blockDim.y*gridDim.y;

  // select input/output plane
  gradOutput = gradOutput + o*output_w*output_h;
  gradInput = gradInput + i*input_w*input_h;
  indices_x = indices_x + o*output_w*output_h;
  indices_y = indices_y + o*output_w*output_h;

  // compute gradInput
  for(yy = yy_start; yy < yy_end; yy+=yy_step) {

    int y_start = (int)floor(float(yy) / output_h * input_h);

    for(xx = xx_start; xx < xx_end; xx+=xx_step) {

      int x_start = (int)floor(float(xx) / output_w * input_w);

      float *ptr_gradInput = gradInput + y_start*input_w + x_start;
      float *ptr_gradOutput = gradOutput + yy*output_w + xx;
      float *ptr_ind_x = indices_x + yy*output_w + xx;
      float *ptr_ind_y = indices_y + yy*output_w + xx;
      float z = *ptr_gradOutput;

      int argmax_x = (*ptr_ind_x)-1;
      int argmax_y = (*ptr_ind_y)-1;

      // atomic add since different threads could update same variable
      atomicAdd(&(ptr_gradInput[argmax_x + argmax_y*input_w]), z);
    }
  }
}

static int cunn_SpatialAdaptiveMaxPooling_updateOutput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");

  long nOutputCols = luaT_getfieldcheckint(L, 1, "W");
  long nOutputRows = luaT_getfieldcheckint(L, 1, "H");

  THCudaTensor *output = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  THCudaTensor *indices = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "indices", "torch.CudaTensor");
  THAssert(THCudaTensor_checkGPU(state, 3, input, output, indices));

  float *indices_data;
  float *output_data;
  float *input_data;

  luaL_argcheck(L, input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch) tensor expected");

  if (input->nDimension == 3) {
    long nInputCols = input->size[2];
    long nInputRows = input->size[1];
    long nInputPlane = input->size[0];

    input = THCudaTensor_newContiguous(state, input);
    input_data = THCudaTensor_data(state, input);

    THCudaTensor_resize3d(state, output, nInputPlane, nOutputRows, nOutputCols);
    THCudaTensor_resize4d(state, indices, 2, nInputPlane, nOutputRows, nOutputCols);

    indices_data = THCudaTensor_data(state, indices);
    output_data = THCudaTensor_data(state, output);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane,yblocks);
    dim3 threads(32,8);

    // run maxpool kernel
    adaptivemaxpool <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (input_data, output_data,
                                   indices_data+nInputPlane*nOutputCols*nOutputRows, indices_data,
                                   nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols);
  } else {
    long nInputCols = input->size[3];
    long nInputRows = input->size[2];
    long nInputPlane = input->size[1];
    long nbatch = input->size[0];

    input = THCudaTensor_newContiguous(state, input);
    input_data = THCudaTensor_data(state, input);

    THCudaTensor_resize4d(state, output, nbatch, nInputPlane, nOutputRows, nOutputCols);
    THCudaTensor_resize5d(state, indices, 2, nbatch, nInputPlane, nOutputRows, nOutputCols);

    indices_data = THCudaTensor_data(state, indices);
    output_data = THCudaTensor_data(state, output);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane*nbatch,yblocks);
    dim3 threads(32,8);

    // run maxpool kernel
    adaptivemaxpool <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (input_data, output_data,
                                   indices_data+nbatch*nInputPlane*nOutputCols*nOutputRows, indices_data,
                                   nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols);
  }

  // clean
  THCudaTensor_free(state, input);

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in SpatialAdaptiveMaxPooling.updateOutput: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
  return 1;
}

static int cunn_SpatialAdaptiveMaxPooling_updateGradInput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");

  bool atomic = true; // suboptimal, but without atomic it doesn't pass the tests

  THCudaTensor *gradInput = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  THCudaTensor *indices = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "indices", "torch.CudaTensor");
  THAssert(THCudaTensor_checkGPU(state, 4, input, indices, gradOutput, gradInput));

  float *indices_data;
  float *gradInput_data;
  float *gradOutput_data;

  if (input->nDimension == 3) {
    long nInputCols = input->size[2];
    long nInputRows = input->size[1];
    long nInputPlane = input->size[0];
    long nOutputCols = gradOutput->size[2];
    long nOutputRows = gradOutput->size[1];

    //bool atomic = (nInputCols%nOutputCols != 0) || (nInputRows%nOutputRows != 0);

    THCudaTensor_resizeAs(state, gradInput, input);
    THCudaTensor_zero(state, gradInput);

    indices_data = THCudaTensor_data(state, indices);
    gradOutput_data = THCudaTensor_data(state, gradOutput);
    gradInput_data = THCudaTensor_data(state, gradInput);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane,yblocks);
    dim3 threads(32,8);

    if(atomic)
    {
      // run updateGradInput kernel, accumulate gradients atomically
      atomicadaptivemaxgradinput <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (gradInput_data, gradOutput_data,
                                          indices_data+nInputPlane*nOutputCols*nOutputRows, indices_data,
                                          nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols);
    }
    else
    {
      // run updateGradInput kernel
      atomicadaptivemaxgradinput <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (gradInput_data, gradOutput_data,
                                          indices_data+nInputPlane*nOutputCols*nOutputRows, indices_data,
                                          nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols);
    }
  } else {
    long nInputCols = input->size[3];
    long nInputRows = input->size[2];
    long nInputPlane = input->size[1];
    long nbatch = input->size[0];
    long nOutputCols = gradOutput->size[3];
    long nOutputRows = gradOutput->size[2];

    //bool atomic = //(nInputCols%nOutputCols != 0) || (nInputRows%nOutputRows != 0);

    THCudaTensor_resizeAs(state, gradInput, input);
    THCudaTensor_zero(state, gradInput);

    indices_data = THCudaTensor_data(state, indices);
    gradOutput_data = THCudaTensor_data(state, gradOutput);
    gradInput_data = THCudaTensor_data(state, gradInput);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane*nbatch,yblocks);
    dim3 threads(32,8);

    if(atomic)
    {
      // run updateGradInput kernel, accumulate gradients atomically
      atomicadaptivemaxgradinput <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (gradInput_data, gradOutput_data,
                                          indices_data+nbatch*nInputPlane*nOutputCols*nOutputRows, indices_data,
                                          nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols);
    }
    else
    {
      // run updateGradInput kernel, accumulate gradients atomically
      adaptivemaxgradinput <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (gradInput_data, gradOutput_data,
                                          indices_data+nbatch*nInputPlane*nOutputCols*nOutputRows, indices_data,
                                          nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols);
    }
  }

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in SpatialAdaptiveMaxPooling.updateGradInput: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
  return 1;
}

static const struct luaL_Reg cunn_SpatialAdaptiveMaxPooling__ [] = {
  {"SpatialAdaptiveMaxPooling_updateOutput", cunn_SpatialAdaptiveMaxPooling_updateOutput},
  {"SpatialAdaptiveMaxPooling_updateGradInput", cunn_SpatialAdaptiveMaxPooling_updateGradInput},
  {NULL, NULL}
};

static void cunn_SpatialAdaptiveMaxPooling_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_SpatialAdaptiveMaxPooling__, "nn");
  lua_pop(L,1);
}

#undef CUDA_MAX_THREADS
